#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include "ParallelTracer.cuh"
#include "glm/glm.hpp"
#include "Tracer.h"
#include <iostream>
#include "Scene.h"
#include "Tracer.h"
#include <cstdlib>
#include "glm/gtx/perpendicular.hpp"


struct Devim {
	glm::uvec3 *texture;
	int pitch;
	int width;
	int height;
}; 


//next function from http://ray-tracing.ru/articles245.html
__device__ bool RaySphereIntersection2(glm::vec3 &ray_pos, glm::vec3 &ray_dir, glm::vec3 &spos, float r, float& tResult)
{
  //a == 1; // because rdir must be normalized
  glm::vec3 k = ray_pos - spos;
  double b = glm::dot(k, ray_dir);
  double c = glm::dot (k, k) - r*r;
  double d = b*b - c;
 
  if(d >=0)
  {
    double sqrtfd = sqrtf(d);
    // t, a == 1
    double t1 = -b + sqrtfd;
    double t2 = -b - sqrtfd;
 
    double min_t  = glm::min(t1,t2);
    double max_t = glm::max(t1,t2);
 
    double t = (min_t >= 0) ? min_t : max_t;
    tResult = float(t);
    return (t > 0);
  }
  return false;
}

__device__ 	bool DiskIntersection(glm::vec3 &raypos, glm::vec3 &dir, float &t)
{
	bool ans = true;
	ans = abs(dir.z) >= 0.0000000001;
	if (ans){
		t = float(-raypos.z / dir.z);
		ans = ans & t >=0;
	}
	return ans;
}

__device__ SRay MakeRay(CTracer *t, glm::uvec2 pixelPos)
{
	int xRes = t->m_camera.m_resolution.x;
	int yRes = t->m_camera.m_resolution.y;
	float coef = float(xRes) / (2 * tan(t->m_camera.m_viewAngle.x / 2 ));
	glm::vec3 ViewDir = t->m_camera.m_forward *coef;
	coef = float(xRes) ;
	glm::vec3 Right = t->m_camera.m_right * coef;
	coef = float(yRes);
	glm::vec3 Up = t->m_camera.m_up * coef;
	float one = (pixelPos.x + 0.5)/xRes - 0.5;
	float two = (pixelPos.y + 0.5)/yRes - 0.5;

	glm::vec3 dir = glm::normalize(ViewDir + Right * one + Up * two);
	glm::vec3 start = t->m_camera.m_pos;
	SRay ray;
	ray.m_dir = dir;
	ray.m_start = start;
  return ray;
}

__device__ glm::uvec3 getpix(Devim pic, int x, int y)
{
	return pic.texture[x * pic.width + y] ;
} 


__device__ float len (glm::vec3 v){
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ glm::vec3 TraceRay(CTracer *th, SRay ray, Devim backtext, Devim disktext)
{
	glm::vec3 answer(1,0,0);
	float c_light = 300000000;
	float Gravity_const = 6.67408e-11;
	float llastrad, rad = glm::length(ray.m_start);
	glm::vec3 lastpos = ray.m_start;
	glm::vec3 lastdir = ray.m_dir;
	float dt = 0.00002;
	//glm::vec3 C(c_light, c_light, c_light);
	//glm::vec3 G(Gravity_const, Gravity_const, Gravity_const);
	float G = Gravity_const;
	float C = c_light;
	float M = th->m_pScene->bh.mass; 
	for (int i = 0; i < 100000; i++){
		
		llastrad = rad;
		lastpos = ray.m_start;
		lastdir = ray.m_dir;

		// compute new rate
		glm::vec3 a = -glm::normalize(ray.m_start) * G * M / glm::dot(ray.m_start, ray.m_start);
		// we should use only normal component
		glm::vec3 normal = glm::perp(a, ray.m_dir);
		//compute new dir
		ray.m_dir = glm::normalize(ray.m_dir * C + normal * dt) ;
		//compute peremeshenie with blackhole
		glm::vec3 dist = C * lastdir * dt + normal * dt * dt / 2.0f;
		ray.m_start += dist;
		float rad = glm::length(ray.m_start);
		float t;

		if (DiskIntersection(lastpos, glm::normalize(dist), t)){
			glm::vec3 hlp = lastpos + glm::normalize(dist) * t;
			if (t < glm::length(dist) && glm::length(hlp) < th->m_pScene->ad.radius){ 
				float x = hlp.x, y = hlp.y;
				int m, n;
				m	= (x / (th->m_pScene->ad.radius * 2) + 0.5) * disktext.width;
				n = (y / ( th->m_pScene->ad.radius * 2) + 0.5) * disktext.height;
				glm::uvec3 asd = getpix(disktext, m, n);
				return glm::vec3(asd.x / 255.0f, asd.y / 255.0f, asd.z / 255.0f);
			}
		}  
		if (RaySphereIntersection2(lastpos, glm::normalize(dist), glm::vec3(0,0,0), th->m_pScene->bh.radius, t ))
		{
			if (t < glm::length(dist)) {
				return glm::vec3(0,0,0); 
			}
		} 
		if (rad > llastrad && glm::length(ray.m_dir - lastdir) < 0.001 ){
			break;
		}
		dt = c_light / (1000 * glm::length(a));
	}

	float mPi = 3.14159265359f;
	float x, y, z;
	x = ray.m_dir.x; y = ray.m_dir.y; z = ray.m_dir.z;
	float fi = atan2f(x, y);
	float teta = asinf(z);
	fi = (fi + mPi) * (float(backtext.width) - 1) / (2 * mPi);
	teta = (teta + mPi / 2)  *( float( backtext.height ) - 1) / (mPi);
	glm::uvec3 t = getpix(backtext, int(teta), int(fi));
	return glm::vec3(float(t.x) / 255.0, float(t.y) / 255.0, float(t.z) / 255.0); 
}

__global__ void iter(CTracer *th, glm::vec3 *cam, int xres, int yres,
					Devim b, Devim d)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if (k < xres * yres)
	{
		 SRay ray = MakeRay(th, glm::uvec2(k % xres,  k / xres));
		 cam[k] = TraceRay(th, ray, b, d);
	}
}


void help(CTracer *th, int xRes, int yRes, glm::vec3 *pic) {
	
	CScene *sc;
	glm::vec3 *cam;
	hipMalloc((void **)&cam, xRes * yRes * sizeof(glm::vec3));
	hipMalloc((void **) &sc, sizeof(CScene));
	hipMemcpy(sc, th->m_pScene, sizeof(CScene), hipMemcpyHostToDevice);
	
	Devim d;
	Devim b;
	
	d.pitch = th->m_pScene->ad.texture->pitch;
	d.width = th->m_pScene->ad.texture->im->GetWidth();
	d.height = th->m_pScene->ad.texture->im->GetHeight();
	glm::uvec3 *tmp = (glm::uvec3 *) malloc(sizeof(glm::uvec3) * d.width * d.height);
	for (int i = 0; i < d.height; i++){
		for (int j = 0; j < d.width; j++){
			tmp[i * d.width + j] = th->m_pScene->ad.texture->getpix(i,j);
		}
	}
	glm::uvec3 *disktext;

	hipMalloc((void **)&disktext, sizeof(glm::uvec3) * d.width * d.height);

	hipMemcpy(disktext, tmp, sizeof(glm::uvec3) * d.width * d.height, hipMemcpyHostToDevice);
	d.texture = disktext;
	free(tmp);


	b.pitch = th->m_pScene->fone->pitch;
	b.width = th->m_pScene->fone->im->GetWidth();
	b.height = th->m_pScene->fone->im->GetHeight();

	tmp = (glm::uvec3 *) malloc(sizeof(glm::uvec3) * b.width * b.height);
	for (int i = 0; i < b.height; i++){
		for (int j = 0; j < b.width; j++){
			tmp[i * b.width + j] = th->m_pScene->fone->getpix(i,j);
		}
	}
	
	glm::uvec3 *backtext;
	hipMalloc((void **)&backtext, sizeof(glm::uvec3) * b.width * b.height);
	hipMemcpy(backtext, tmp, b.width * b.height * sizeof(glm::uvec3), hipMemcpyHostToDevice);
	free(tmp);
	b.texture = backtext;

	th->m_pScene = sc;
	CTracer *q;
	hipMalloc((void **)&q, sizeof(CTracer));

	hipMemcpy(cam, pic, sizeof(glm::vec3) * xRes * yRes, hipMemcpyHostToDevice);

	hipMemcpy(q, th, sizeof(CTracer), hipMemcpyHostToDevice);
	//iter<<<gridDim, blockDim>>>(q, cam, xRes, yRes, alisa);
	iter<<<ceilf(xRes * yRes / 512.0f), 512>>>(q, cam, xRes, yRes, b, d);
	hipMemcpy(pic, cam, sizeof(glm::vec3) * xRes * yRes, hipMemcpyDeviceToHost);
}